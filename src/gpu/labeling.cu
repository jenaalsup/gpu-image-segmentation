#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "labeling.cuh"

static constexpr int THREADS_PER_BLOCK = 256;

__global__
void initializeLabels(const unsigned char* input, int* labels, int width, int height) {
    // each thread is responsible for one pixel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;
    if (idx >= total_pixels) return;
    // label each foreground pixel with a unique ID
    labels[idx] = (input[idx] == 255) ? idx + 1 : 0; 
}

// iteratively propgate the smallest pixel in a neighborhood to all of its neighbors
__global__
void propagateLabels(const unsigned char* input, int* labels, int width, int height, bool* changed) {
    // each thread is responsible for checking the 8 connected neighbors of a pixel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;
    if (idx >= total_pixels) return;

    int x = idx % width;
    int y = idx / width;

    if (input[idx] != 255) return;

    int label = labels[idx];
    int min_label = label;

    // 8-neighbor directions
    int dx[] = {-1, 0, 1, -1, 1, -1, 0, 1};
    int dy[] = {-1, -1, -1, 0, 0, 1, 1, 1};

    for (int k = 0; k < 8; k++) {
        int nx = x + dx[k];
        int ny = y + dy[k];
        if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
            int n_idx = ny * width + nx;
            if (input[n_idx] == 255 && labels[n_idx] < min_label && labels[n_idx] > 0) {
                min_label = labels[n_idx];
            }
        }
    }

    if (min_label < label) {
        labels[idx] = min_label;
        *changed = true;
    }
}

__global__
void compressLabels(int* labels, int width, int height) {
    // each thread is responsible for one pixel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;
    if (idx >= total_pixels) return;

    // go up the chain of labels until you find the root which originally had the smallest label
    if (labels[idx] > 0) {
        int root = labels[idx];
        while (labels[root - 1] != root) { // subtract 1 because labels are 1-based but array indices are 0-based
            root = labels[root - 1];
        }
        labels[idx] = root;
    }
}

void runLabeling(const unsigned char* h_input, int* h_labels, int width, int height) {
    size_t img_size = width * height * sizeof(unsigned char);
    size_t label_size = width * height * sizeof(int);

    unsigned char* d_input;
    int* d_labels;
    bool* d_changed;
    bool h_changed;

    hipMalloc(&d_input, img_size);
    hipMalloc(&d_labels, label_size);
    hipMalloc(&d_changed, sizeof(bool));
    hipMemcpy(d_input, h_input, img_size, hipMemcpyHostToDevice);

    int total_pixels = width * height;
    int num_blocks = (total_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    initializeLabels<<<num_blocks, THREADS_PER_BLOCK>>>(d_input, d_labels, width, height);
    hipDeviceSynchronize();

    // repeat propogateLabels until there are no more label changes
    do {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);
        propagateLabels<<<num_blocks, THREADS_PER_BLOCK>>>(d_input, d_labels, width, height, d_changed);
        hipDeviceSynchronize();
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
    } while (h_changed);

    // compress labels such that every pxel in the same component has the exact same label 
    compressLabels<<<num_blocks, THREADS_PER_BLOCK>>>(d_labels, width, height);
    hipDeviceSynchronize();

    // copy labels back to host
    hipMemcpy(h_labels, d_labels, label_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_labels);
    hipFree(d_changed);
}
