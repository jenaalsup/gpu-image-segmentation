#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <numeric>
#include <vector>
#include <algorithm>
#include "threshold.cuh"

static constexpr int THREADS_PER_BLOCK = 256;

__global__
void histogramKernel(const unsigned char* input, int* hist, int width, int height, int ignore_below) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;
    if (idx >= total_pixels) return;

    unsigned char val = input[idx];
    if (val >= ignore_below) atomicAdd(&hist[val], 1);
}

__global__
void binarizeKernel(const unsigned char* input, unsigned char* output, int width, int height, int threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;
    if (idx >= total_pixels) return;
    output[idx] = input[idx] > threshold ? 255 : 0;
}

int findOtsuThreshold(const unsigned char* d_input, int width, int height) {
    const int ignore_below = 15;
    int* d_hist;
    hipMalloc(&d_hist, 256 * sizeof(int));
    hipMemset(d_hist, 0, 256 * sizeof(int));

    int total_pixels = width * height;
    int num_blocks = (total_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    histogramKernel<<<num_blocks, THREADS_PER_BLOCK>>>(d_input, d_hist, width, height, ignore_below);
    hipDeviceSynchronize();

    std::vector<int> h_hist(256);
    hipMemcpy(h_hist.data(), d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_hist);

    int total = std::accumulate(h_hist.begin() + ignore_below, h_hist.end(), 0);
    if (total == 0) return 0;  // no valid pixels

    double sum_all = 0.0;
    for (int i = 0; i < 256; i++) sum_all += i * h_hist[i];

    double sum_bg = 0.0;
    int weight_bg = 0;
    double max_var = 0.0;
    int best_thresh = ignore_below;

    for (int t = ignore_below; t < 256; t++) {
        weight_bg += h_hist[t];
        if (weight_bg == 0) continue;
        int weight_fg = total - weight_bg;
        if (weight_fg == 0) break;

        sum_bg += t * h_hist[t];
        double mean_bg = sum_bg / weight_bg;
        double mean_fg = (sum_all - sum_bg) / weight_fg;

        double var = weight_bg * weight_fg * (mean_bg - mean_fg) * (mean_bg - mean_fg);
        if (var > max_var) {
            max_var = var;
            best_thresh = t;
        }
    }

    return best_thresh;
}

void binarizeImage(const unsigned char* d_input, unsigned char* d_output, int width, int height, int threshold) {
    int total_pixels = width * height;
    int num_blocks = (total_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    binarizeKernel<<<num_blocks, THREADS_PER_BLOCK>>>(d_input, d_output, width, height, threshold);
    hipDeviceSynchronize();
}

void runThresholding(const unsigned char* h_input, unsigned char* h_output, int width, int height) {
    size_t img_size = width * height * sizeof(unsigned char);

    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, img_size);
    hipMalloc(&d_output, img_size);
    hipMemcpy(d_input, h_input, img_size, hipMemcpyHostToDevice);

    int threshold = findOtsuThreshold(d_input, width, height);
    binarizeImage(d_input, d_output, width, height, threshold);
    hipMemcpy(h_output, d_output, img_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
